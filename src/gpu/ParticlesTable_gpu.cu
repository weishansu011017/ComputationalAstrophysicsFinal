#include "hip/hip_runtime.h"
#include "ParticlesTable.hpp"
#include <hip/hip_runtime.h> 
#include <cstdio>

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost

#define CUDA_CHECK(call)                                                     \
    do {                                                                     \
        hipError_t err = call;                                              \
        if (err != hipSuccess) {                                            \
            fprintf(stderr, "CUDA error %s (%d) at %s:%d\n",                 \
                    hipGetErrorString(err), err, __FILE__, __LINE__);       \
            std::abort();                                                    \
        }                                                                    \
    } while (0)

// Device-only function
// --- Direct N-body  ---
__global__ void dirnbody_kernel(const float* __restrict x,
                                const float* __restrict y,
                                const float* __restrict z,
                                const float* __restrict h,
                                const float* __restrict m,
                                float* __restrict ax,
                                float* __restrict ay,
                                float* __restrict az,
                                float* __restrict U,
                                int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float xi = x[i], yi = y[i], zi = z[i], hi = h[i];
    float axt = 0.f, ayt = 0.f, azt = 0.f, Ut = 0.f;

    for (int j = 0; j < N; ++j) {
        if (i == j) continue;
        float dx = x[j] - xi;
        float dy = y[j] - yi;
        float dz = z[j] - zi;
        float dr2 = dx*dx + dy*dy + dz*dz + 0.5f*(hi*hi + h[j]*h[j]);
        float invr = rsqrtf(dr2);       
        float invr3 = invr * invr * invr;

        float mjinvr3 = m[j] * invr3;
        axt += mjinvr3 * dx;
        ayt += mjinvr3 * dy;
        azt += mjinvr3 * dz;
        Ut  -= m[i] * m[j] * invr;
    }
    ax[i] = axt;  ay[i] = ayt;  az[i] = azt;  U[i] = Ut;
}

__global__ void dirnbody_2D_kernel(const float* __restrict x,
                                const float* __restrict y,
                                const float* __restrict h,
                                const float* __restrict m,
                                float* __restrict ax,
                                float* __restrict ay,
                                float* __restrict U,
                                int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float xi = x[i], yi = y[i], hi = h[i];
    float axt = 0.f, ayt = 0.f, Ut = 0.f;

    for (int j = 0; j < N; ++j) {
        if (i == j) continue;
        float dx = x[j] - xi;
        float dy = y[j] - yi;
        float dr2 = dx*dx + dy*dy + 0.5f*(hi*hi + h[j]*h[j]);
        float invr = rsqrtf(dr2);       
        float invr3 = invr * invr * invr;

        float mjinvr3 = m[j] * invr3;
        axt += mjinvr3 * dx;
        ayt += mjinvr3 * dy;
        Ut  -= m[i] * m[j] * invr;
    }
    ax[i] = axt;  ay[i] = ayt;  U[i] = Ut;
}

// --- BH Tree ---
__global__ void BHtree_kernel(const float* __restrict x,
                                const float* __restrict y,
                                const float* __restrict z,
                                const float* __restrict h,
                                const float* __restrict m,
                                float* __restrict ax,
                                float* __restrict ay,
                                float* __restrict az,
                                float* __restrict U,
                                const OctNode* __restrict nodes,
                                const int* __restrict order,
                                int root_idx,
                                float theta,
                                int N)
{
    // Get idx
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Construct a stack to mimic the recursion
    const int MAX_STACK = 256;
    int stack[MAX_STACK];
    int sp = 0;                     // Current number of stack
    stack[sp++] = root_idx;

    // Get current quantities
    float xi = x[idx];
    float yi = y[idx];
    float zi = z[idx];
    float hi = h[idx];
    float mi = m[idx];

    float axi = 0.0f, ayi = 0.0f, azi = 0.0f, Ui = 0.0f;

    while (sp > 0) {
        int nidx = stack[--sp];  // Get node index
        const OctNode& node = nodes[nidx];

        if (node.pcount == 0) continue;

        if (node.isLeaf()) {
            for (int p = 0; p < node.pcount; ++p) {
                int j = order[node.ParticlesLocateidx + p];
                if (j == idx) continue;

                float dx = x[j] - xi;
                float dy = y[j] - yi;
                float dz = z[j] - zi;
                float r2 = dx*dx + dy*dy + dz*dz + 0.5f * (hi*hi + h[j]*h[j]);
                float invr = rsqrtf(r2);
                float invr3 = invr * invr * invr;
                axi += m[j] * dx * invr3;
                ayi += m[j] * dy * invr3;
                azi += m[j] * dz * invr3;
                Ui -= mi * m[j] * invr;
            }
            continue;
        }

        float dx = node.COMx - xi;
        float dy = node.COMy - yi;
        float dz = node.COMz - zi;
        float r2 = dx*dx + dy*dy + dz*dz;

        if (r2 > 1e-8f && node.cellsize() * rsqrtf(r2) < theta) {
            r2 += hi*hi;
            float invr = rsqrtf(r2);
            float invr3 = invr * invr * invr;
            float mInvr3 = node.Mtot * invr3;
            axi += mInvr3 * dx;
            ayi += mInvr3 * dy;
            azi += mInvr3 * dz;
            Ui -= mi * node.Mtot * invr;
        } else {
            for (int q = 0; q < 4; ++q) {
                int cidx = node.children[q];
                if (cidx >= 0 && sp < MAX_STACK){
                    stack[sp++] = cidx;
                }
            }
        }
    }
    // Store the acc into GPU array
    ax[idx] = axi;
    ay[idx] = ayi;
    az[idx] = azi;
    U[idx]  = Ui;
}

__global__ void BHtree_2D_kernel(const float* __restrict x,
                                const float* __restrict y,
                                const float* __restrict h,
                                const float* __restrict m,
                                float* __restrict ax,
                                float* __restrict ay,
                                float* __restrict U,
                                const QuadNode* __restrict nodes,
                                const int* __restrict order,
                                int root_idx,
                                float theta,
                                int N)
{
    // Get idx
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Construct a stack to mimic the recursion
    const int MAX_STACK = 128;
    int stack[MAX_STACK];
    int sp = 0;                     // Current number of stack
    stack[sp++] = root_idx;

    // Get current quantities
    float xi = x[idx];
    float yi = y[idx];
    float hi = h[idx];
    float mi = m[idx];

    float axi = 0.0f, ayi = 0.0f, Ui = 0.0f;

    while (sp > 0) {
        int nidx = stack[--sp];  // Get node index
        const QuadNode& node = nodes[nidx];

        if (node.pcount == 0) continue;

        if (node.isLeaf()) {
            for (int p = 0; p < node.pcount; ++p) {
                int j = order[node.ParticlesLocateidx + p];
                if (j == idx) continue;

                float dx = x[j] - xi;
                float dy = y[j] - yi;
                float r2 = dx*dx + dy*dy + 0.5f * (hi*hi + h[j]*h[j]);
                float invr = rsqrtf(r2);
                float invr3 = invr * invr * invr;
                axi += m[j] * dx * invr3;
                ayi += m[j] * dy * invr3;
                Ui -= mi * m[j] * invr;
            }
            continue;
        }

        float dx = node.COMx - xi;
        float dy = node.COMy - yi;
        float r2 = dx*dx + dy*dy;

        if (r2 > 1e-8f && node.cellsize() * rsqrtf(r2) < theta) {
            r2 += hi*hi;
            float invr = rsqrtf(r2);
            float invr3 = invr * invr * invr;
            float mInvr3 = node.Mtot * invr3;
            axi += mInvr3 * dx;
            ayi += mInvr3 * dy;
            Ui -= mi * node.Mtot * invr;
        } else {
            for (int q = 0; q < 4; ++q) {
                int cidx = node.children[q];
                if (cidx >= 0 && sp < MAX_STACK){
                    stack[sp++] = cidx;
                }
            }
        }
    }
    // Store the acc into GPU array
    ax[idx] = axi;
    ay[idx] = ayi;
    U[idx]  = Ui;
}



// --- Kick ---
__global__ void kick_kernel(float* __restrict vx,
                            float* __restrict vy,
                            float* __restrict vz,
                            const float* __restrict ax,
                            const float* __restrict ay,
                            const float* __restrict az,
                            const float* __restrict dt,
                            float scale,
                            int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    float dt_i = dt[i];
    float fac  = scale * dt_i;
    vx[i] += fac * ax[i];
    vy[i] += fac * ay[i];
    vz[i] += fac * az[i];
}

__global__ void kick_2D_kernel( float* __restrict vx,
                                float* __restrict vy,
                                const float* __restrict ax,
                                const float* __restrict ay,
                                const float* __restrict dt,
                                float scale,
                                int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    float dt_i = dt[i];
    float fac  = scale * dt_i;
    vx[i] += fac * ax[i];
    vy[i] += fac * ay[i];
}

// --- Drift ---
__global__ void drift_kernel(float* __restrict x,
                             float* __restrict y,
                             float* __restrict z,
                             const float* __restrict vx,
                             const float* __restrict vy,
                             const float* __restrict vz,
                             const float* __restrict dt,
                             float scale,
                             int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    float dt_i = dt[i];
    float fac  = scale * dt_i;
    x[i] += fac * vx[i];
    y[i] += fac * vy[i];
    z[i] += fac * vz[i];
}

__global__ void drift_2D_kernel(float* __restrict x,
                                float* __restrict y,
                                const float* __restrict vx,
                                const float* __restrict vy,
                                const float* __restrict dt,
                                float scale,
                                int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    float dt_i = dt[i];
    float fac  = scale * dt_i;
    x[i] += fac * vx[i];
    y[i] += fac * vy[i];
}




// Method Definition
void ParticlesTable::device_init() {
    if (gpu_init) return;
    size_t n = static_cast<size_t>(N) * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_x , n)); CUDA_CHECK(hipMalloc(&d_y , n)); CUDA_CHECK(hipMalloc(&d_z , n));
    CUDA_CHECK(hipMalloc(&d_vx, n)); CUDA_CHECK(hipMalloc(&d_vy, n)); CUDA_CHECK(hipMalloc(&d_vz, n));
    CUDA_CHECK(hipMalloc(&d_m , n)); CUDA_CHECK(hipMalloc(&d_h , n)); CUDA_CHECK(hipMalloc(&d_dt, n));
    CUDA_CHECK(hipMalloc(&d_ax, n)); CUDA_CHECK(hipMalloc(&d_ay, n)); CUDA_CHECK(hipMalloc(&d_az, n));
    CUDA_CHECK(hipMalloc(&d_U , n));

    upload_all();  
    gpu_init = true;
}

void ParticlesTable::device_finalize() {
    if (!gpu_init) return;
    hipFree(d_x);  hipFree(d_y);  hipFree(d_z);
    hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
    hipFree(d_m);  hipFree(d_h); hipFree(d_dt);
    hipFree(d_ax); hipFree(d_ay); hipFree(d_az); hipFree(d_U);
    hipFree(d_nodes_2D); hipFree(d_nodes_3D); hipFree(d_order);
    gpu_init = false;
}

void ParticlesTable::upload_all() {
    size_t n=N*sizeof(float);
    hipMemcpyAsync(d_x , x.data() , n, H2D, stream);
    hipMemcpyAsync(d_y , y.data() , n, H2D, stream);
    hipMemcpyAsync(d_z , z.data() , n, H2D, stream);
    hipMemcpyAsync(d_vx, vx.data(), n, H2D, stream);
    hipMemcpyAsync(d_vy, vy.data(), n, H2D, stream);
    hipMemcpyAsync(d_vz, vz.data(), n, H2D, stream);
    hipMemcpyAsync(d_m , m.data() , n, H2D, stream);
    hipMemcpyAsync(d_h , h.data() , n, H2D, stream);
    hipMemcpyAsync(d_dt, dt.data(), n, H2D, stream);
    hipStreamSynchronize(stream);
}

void ParticlesTable::download_state() {           
    size_t n=N*sizeof(float);
    hipMemcpyAsync(x.data() , d_x , n, D2H, stream);
    hipMemcpyAsync(y.data() , d_y , n, D2H, stream);
    hipMemcpyAsync(z.data() , d_z , n, D2H, stream);
    hipMemcpyAsync(vx.data(), d_vx, n, D2H, stream);
    hipMemcpyAsync(vy.data(), d_vy, n, D2H, stream);
    hipMemcpyAsync(vz.data(), d_vz, n, D2H, stream);
    hipMemcpyAsync(_ax.data(),d_ax, n, D2H, stream);
    hipMemcpyAsync(_ay.data(),d_ay, n, D2H, stream);
    hipMemcpyAsync(_az.data(),d_az, n, D2H, stream);
    hipMemcpyAsync(_U.data(), d_U,  n, D2H, stream);

    hipStreamSynchronize(stream);
}

void ParticlesTable::calculate_a_dirnbody_gpu() {
    if (!gpu_init) device_init();
    int grid = (N + block - 1) / block;
    dirnbody_kernel<<<grid, block, 0, stream>>>(
        d_x, d_y, d_z, d_h, d_m,
        d_ax, d_ay, d_az, d_U, N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::calculate_a_dirnbody_2D_gpu() {
    if (!gpu_init) device_init();
    int grid = (N + block - 1) / block;
    dirnbody_2D_kernel<<<grid, block, 0, stream>>>(
        d_x, d_y, d_h, d_m,
        d_ax, d_ay, d_U, N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::calculate_a_BHtree_gpu() {
    // Prevent Tree recorvering
    hipStreamSynchronize(stream);
    
    // Build quadtree
    OctTree tree = buildOctTree();

    // Get root
    const int root = tree.root_idx;

    // Get theta
    const float theta = tree.bhtheta;

    // Preparing sending tree to GPU
    int n_nodes = tree.nodes_list.size();
    int n_order = tree.order.size();

    // Allocate GPU
    // Free pointer first
    if (d_nodes_3D) hipFree(d_nodes_3D);
    if (d_order)    hipFree(d_order);  
    // Nodes
    CUDA_CHECK(hipMalloc(&d_nodes_3D , n_nodes * sizeof(OctNode)));
    // Order
    CUDA_CHECK(hipMalloc(&d_order ,  n_order * sizeof(int)));

    // Upload tree
    hipMemcpyAsync(d_nodes_3D , tree.nodes_list.data() ,  n_nodes * sizeof(OctNode), H2D, stream);
    hipMemcpyAsync(d_order , tree.order.data() ,  n_order * sizeof(int), H2D, stream);



    if (!gpu_init) device_init();
    int grid = (N + block - 1) / block;

    BHtree_kernel<<<grid, block, 0, stream>>>(
        d_x, d_y,  d_z, d_h, d_m,
        d_ax, d_ay, d_az, d_U, d_nodes_3D, d_order,root, theta,  N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::calculate_a_BHtree_2D_gpu() {
    // Prevent Tree recorvering
    hipStreamSynchronize(stream);

    // Build quadtree
    QuadTree tree = buildQuadTree();

    // Get root
    const int root = tree.root_idx;

    // Get theta
    const float theta = tree.bhtheta;

    // Preparing sending tree to GPU
    int n_nodes = tree.nodes_list.size();
    int n_order = tree.order.size();

    // Allocate GPU
    // Free pointer first
    if (d_nodes_2D) hipFree(d_nodes_2D);
    if (d_order)    hipFree(d_order);  
    // Nodes
    CUDA_CHECK(hipMalloc(&d_nodes_2D , n_nodes * sizeof(QuadNode)));
    // Order
    CUDA_CHECK(hipMalloc(&d_order ,  n_order * sizeof(int)));

    // Upload tree
    hipMemcpyAsync(d_nodes_2D , tree.nodes_list.data() ,  n_nodes * sizeof(QuadNode), H2D, stream);
    hipMemcpyAsync(d_order , tree.order.data() ,  n_order * sizeof(int), H2D, stream);



    if (!gpu_init) device_init();
    int grid = (N + block - 1) / block;

    BHtree_2D_kernel<<<grid, block, 0, stream>>>(
        d_x, d_y, d_h, d_m,
        d_ax, d_ay, d_U, d_nodes_2D, d_order,root, theta,  N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::kick_gpu(float scale){
    int grid = (N + block - 1) / block;
    kick_kernel<<<grid, block, 0, stream>>>(
        d_vx, d_vy, d_vz,
        d_ax, d_ay, d_az,
        d_dt, scale, N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::drift_gpu(float scale){
    int grid = (N + block - 1) / block;
    drift_kernel<<<grid, block, 0, stream>>>(
        d_x, d_y, d_z,
        d_vx, d_vy, d_vz,
        d_dt, scale, N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::kick_2D_gpu(float scale){
    int grid = (N + block - 1) / block;
    kick_2D_kernel<<<grid, block, 0, stream>>>(
        d_vx, d_vy,
        d_ax, d_ay,
        d_dt, scale, N);
    CUDA_CHECK(hipGetLastError());
}

void ParticlesTable::drift_2D_gpu(float scale){
    int grid = (N + block - 1) / block;
    drift_2D_kernel<<<grid, block, 0, stream>>>(
        d_x, d_y,
        d_vx, d_vy,
        d_dt, scale, N);
    CUDA_CHECK(hipGetLastError());
}